#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#include "vec3.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "hittable_list.cuh"
#include "camera.cuh"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

#define RANDVEC3 vec3(hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state))

__device__ vec3 random_in_unit_sphere(hiprandState *local_rand_state)
{
    vec3 p;
    do
    {
        p = 2.0f * RANDVEC3 - vec3(1, 1, 1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

__device__ float crossProduct(vec3 a, vec3 b)
{
    return (a.x() * b.y() + a.y() * b.z() + a.z() * b.x());
}

__device__ vec3 color(const ray &r, hittable **world, hiprandState *local_random_state)
{
    ray cur_ray = r;
    float cur_attenuation = 1.0f;
    vec3 curcol = vec3(0.0, 0.0, 0.0);
    const int bounces = 7;
    hit_record path[bounces];
    int hits = 0;
    for (int i = 0; i < bounces; i++)
    {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
        {
            vec3 target = rec.p + rec.normal + (random_in_unit_sphere(local_random_state) * ( 1 - rec.reflect));
            curcol = rec.color * cur_attenuation;
            path[i] = rec;
            cur_attenuation *= 0.5;
            cur_ray = ray(rec.p, target - rec.p);
            hits++;
        }
        else
        {
             vec3 unit_direction = unit_vector(cur_ray.direction());
             float t = 0.5f * (unit_direction.y() + 1.0f);
            //  vec3 c = (1.0f - t) * vec3(0.1, 0.0, 0.3) + t * vec3(0.5, 0.7, 1.0);
             vec3 c = (1.0f - t) * vec3(0.5, 0.2, 0.1) + t * vec3(0.2, 0.2, 0.2);
            // return curcol + cur_attenuation * c;
            hit_record hr = hit_record();
            hr.color = c;
            hr.luminance = 1;
            path[i] = hr;
            hits++;
            break;
        }
    }

    vec3 color = vec3(0.0, 0.0, 0.0);
    for (int i = hits - 1; i >= 0; i--)
    {
            color = (vec3(color.x() * path[i].color.x(), color.y() * path[i].color.y(), color.z() * path[i].color.z()) * path[i].reflect) + path[i].color * path[i].luminance;// * path[i].reflect;        
    }

    return color;
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hittable **world, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x + i;

    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);

    for (int s = 0; s < ns; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world, &local_rand_state);
    }

    fb[pixel_index] = col / float(ns);
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *(d_list) = new sphere(vec3(0, 0, -1.5), 0.5, vec3(1.0, 0, 0), 0.5, 0.8);               //red
        *(d_list + 1) = new sphere(vec3(0.4, 0.0, -1.0), 0.2, vec3(1.0, 1.0, 1.0), 0.0, 1.0);   //mirror
        *(d_list + 2) = new sphere(vec3(0.5, 0.0, -0.5), 0.1, vec3(0.2, 1.0, 0.3), 0.0, 1.0);   //Green mirror
        *(d_list + 3) = new sphere(vec3(-0.2, -0.1, -0.9), 0.2, vec3(0.0, 0.2, 0.5), 0.0, 0.5); //blue
        *(d_list + 4) = new sphere(vec3(-0.0, -0.1, -0.6), 0.1, vec3(1.0, 1.0, 1.0), 5.0, 0.0); //white / light source
        *(d_list + 5) = new sphere(vec3(0, -100.5, -1), 100, vec3(0.5, 0.5, 0.5), 0.0, 0.5);    //big / floor
        *(d_list + 6) = new sphere(vec3(0.4, 7, -1), 5, vec3(1.0, 1.0, 1.0), 2.0, 1.0);    //light source 2
        *d_world = new hittable_list(d_list, 7);
        *d_camera = new camera();
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
    delete *d_camera;
}

int main()
{
    int nx = 1200;
    int ny = 600;
    int ns = 1000; // Number of samples
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // make our world of hitables
    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n"
              << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}