#include "hip/hip_runtime.h"
#include "gui.cuh"
#include "cuda/gpu_render.cuh"
#include "cpp/cpu_render.hh"
#include <thread>

int main()
{
    glfwSetErrorCallback(errorCallback);

    int window_height = 1080;
    auto window = create_window(window_height, 16.0 / 9.0);
    if (!window)
    {
        printf("Creation of window failed!");
        glfwTerminate();
        return 1;
    }

    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, frameBufferSizeCallback);
    glfwSetKeyCallback(window, keyCallback);

    GLenum err = glewInit();
    if (err != GLEW_OK)
    {
        printf("Init of glew failed! %s\n", glewGetErrorString(err));
    }

    // Setup Dear ImGui context
    ImGuiIO &io = setupImGUI(window);

    // Our state
    ImVec4 clear_color = ImVec4(0.0f, 0.1f, 0.2f, 1.00f);

    GLuint image;

    float image_aspect_ratio = 16.0f / 9.0f;
    double last_render_time = 0.0;
    while (!glfwWindowShouldClose(window))
    {
        handleEvents(window);
        glfwMakeContextCurrent(window);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w, clear_color.w);

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        {
            const ImGuiViewport *viewport = ImGui::GetMainViewport();
            ImGui::SetNextWindowPos(viewport->WorkPos);
            ImGui::SetNextWindowSize(viewport->WorkSize);
            ImGui::SetNextWindowViewport(viewport->ID);
            ImGui::PushStyleVar(ImGuiStyleVar_WindowRounding, 0.0f);
            ImGui::PushStyleVar(ImGuiStyleVar_WindowBorderSize, 0.0f);

            ImGuiWindowFlags window_flags = ImGuiWindowFlags_NoBackground | ImGuiWindowFlags_NoDocking;
            window_flags |= ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoCollapse | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoMove;
            window_flags |= ImGuiWindowFlags_NoBringToFrontOnFocus | ImGuiWindowFlags_NoNavFocus;

            static ImGuiDockNodeFlags dockspace_flags = ImGuiDockNodeFlags_AutoHideTabBar;
            dockspace_flags |= ImGuiDockNodeFlags_NoDockingInCentralNode | ImGuiDockNodeFlags_PassthruCentralNode;

            ImGui::PushStyleVar(ImGuiStyleVar_WindowPadding, ImVec2(0.0f, 0.0f));
            ImGui::Begin("DockSpace", reinterpret_cast<bool *>(true), window_flags);
            ImGui::PopStyleVar();
            ImGui::PopStyleVar(2);

            // Submit the DockSpace
            ImGuiID dockspace_id = ImGui::GetID("Dockspace");
            ImGui::DockSpace(dockspace_id, ImVec2(0.0f, 0.0f), dockspace_flags);

            ImGui::End();
        }
        {
            if (show_render)
            {
                const ImGuiViewport *viewport = ImGui::GetMainViewport();
                ImGui::SetNextWindowPos(viewport->WorkPos);
                ImGui::SetNextWindowSize(viewport->WorkSize);
                ImGui::SetNextWindowViewport(viewport->ID);
                ImGui::PushStyleVar(ImGuiStyleVar_WindowRounding, 0.0f);
                ImGui::PushStyleVar(ImGuiStyleVar_WindowBorderSize, 0.0f);
                ImGui::PushStyleVar(ImGuiStyleVar_WindowPadding, ImVec2(0.0f, 0.0f));

                ImGuiWindowFlags window_flags = ImGuiWindowFlags_NoDocking | ImGuiWindowFlags_NoTitleBar;
                window_flags |= ImGuiWindowFlags_NoCollapse | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoMove;
                window_flags |= ImGuiWindowFlags_NoBringToFrontOnFocus | ImGuiWindowFlags_NoNavFocus;

                ImGui::Begin("Image", reinterpret_cast<bool *>(true), window_flags);
                ImGui::PopStyleVar();
                ImGui::PopStyleVar(2);

                ImVec2 pos = ImGui::GetCursorScreenPos();
                ImVec2 ws = ImGui::GetContentRegionAvail();
                int max_x = ws.y * image_aspect_ratio;
                int centering = (ws.x - max_x) / 2;
                ImGui::GetWindowDrawList()->AddImage(
                    reinterpret_cast<ImTextureID>(image),
                    ImVec2(pos.x + centering, pos.y),
                    ImVec2(pos.x + max_x + centering, ws.y + pos.y),
                    ImVec2(0, 1), ImVec2(1, 0));
                ImGui::End();
            }
        }
        {
            ImGui::Begin("Renderer");
            enum Aspect_Ratio
            {
                AR43,
                AR169,
                Ratio_COUNT
            };
            static int ar = AR169;
            const double aspect_ratios[Ratio_COUNT] = {4.0 / 3.0, 16.0 / 9.0};
            const char *aspect_ratio_names[Ratio_COUNT] = {"4:3", "16:9"};
            const char *aspect_ratio = (ar >= 0 && ar < Ratio_COUNT) ? aspect_ratio_names[ar] : "Unknown";

            enum Image_Height
            {
                IW720P,
                IW1080P,
                IW1440P,
                IW4K,
                Height_COUNT
            };
            static int ih = IW1080P;
            const int image_heights[Height_COUNT] = {720, 1080, 1440, 2160};
            const char *height = (ih >= 0 && ih < Height_COUNT) ? std::to_string(image_heights[ih]).c_str() : "Unknown";

            if (ImGui::CollapsingHeader("Image Options", ImGuiTreeNodeFlags_DefaultOpen))
            {
                ImGui::SliderInt("Aspect Ratio", &ar, 0, Ratio_COUNT - 1, aspect_ratio);
                ImGui::SliderInt("Image Height", &ih, 0, Height_COUNT - 1, height);
            }

            enum Render_Method
            {
                CPU,
                GPU,
                OPTION_COUNT
            };
            static int rm = CPU;
            const bool render_methods[OPTION_COUNT] = {false, true};
            const char *render_method_names[OPTION_COUNT] = {"CPU", "GPU"};
            const char *render_method_name = (ar >= 0 && ar < OPTION_COUNT) ? render_method_names[rm] : "Unknown";

            enum SPP
            {
                SPP1,
                SPP10,
                SPP50,
                SPP100,
                SPP500,
                SPP1000,
                SPP_COUNT
            };
            static int spp = SPP10;
            const int spp_values[SPP_COUNT] = {1, 10, 50, 100, 500, 1000};

            enum Depth
            {
                D1,
                D10,
                D25,
                D50,
                DEPTH_COUNT
            };
            static int depth = D10;
            const int depth_values[DEPTH_COUNT] = {1, 10, 25, 50};

            static int cpu_count = std::thread::hardware_concurrency();
            static bool render_on_device = true;
            if (ImGui::CollapsingHeader("Render Options", ImGuiTreeNodeFlags_DefaultOpen))
            {
                ImGui::SliderInt("Render Method", &rm, 0, OPTION_COUNT - 1, render_method_name);
                const char *spp_name = (spp >= 0 && spp < SPP_COUNT) ? std::to_string(spp_values[spp]).c_str() : "Unknown";
                ImGui::SliderInt("Samples per Pixel", &spp, 0, SPP_COUNT - 1, spp_name);
                const char *depth_name = (depth >= 0 && depth < DEPTH_COUNT) ? std::to_string(depth_values[depth]).c_str() : "Unknown";
                ImGui::SliderInt("Max Depth", &depth, 0, DEPTH_COUNT - 1, depth_name);
                ImGui::SliderInt("CPU Cores", &cpu_count, 1, std::thread::hardware_concurrency());
            }

            static int fov = 20;
            static int look_from[3] = {13, 2, 3};
            static int look_at[3] = {0, 0, 0};
            static float defocus_angle = 0.6f;
            if (ImGui::CollapsingHeader("Camera Options", ImGuiTreeNodeFlags_DefaultOpen))
            {
                ImGui::SliderInt("FOV", &fov, 0, 90);
                ImGui::InputInt3("Camera Position", look_from);
                ImGui::InputInt3("Focal Point", look_at);
                ImGui::SliderFloat("Defocus Angle", &defocus_angle, 0, 1.0, "%.1f");
            }
            ImGui::NewLine();

            ImVec2 sz = ImVec2(ImGui::GetWindowWidth() * 0.3f, 0.0f);
            if (ImGui::Button("Render", sz))
            {
                image_aspect_ratio = aspect_ratios[ar];
                render_on_device = render_methods[rm];
                point cam_pos = {look_from[0], look_from[1], look_from[2]};
                point focal_point = {look_at[0], look_at[1], look_at[2]};

                if (render_on_device)
                    gpu_render(image_heights[ih], aspect_ratios[ar], spp_values[spp], depth_values[depth], cam_pos, focal_point, fov, defocus_angle, last_render_time);
                else
                    cpu_render(image_heights[ih], aspect_ratios[ar], spp_values[spp], depth_values[depth], cam_pos, focal_point, fov, defocus_angle, last_render_time);
                // void cpu_render(double _aspect_ratio, int _image_height, int _samples_per_pixel, int _max_depth, double _vfov, point _cam_pos, point _focal_point, double _aperture);

                image = render_image(std::ceil(image_heights[ih] * aspect_ratios[ar]), image_heights[ih]);
                show_render = true;
            }
            if (last_render_time > 0)
            {
                // ImGui::SameLine();
                ImGui::Text("Last render: %.3fs | %dx%d", last_render_time, image_heights[ih],
                            (int)std::ceil(image_heights[ih] * aspect_ratios[ar]));
            }
            ImGui::End();
        }

        // Rendering
        ImGui::Render();

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        // Update and Render additional Platform Windows
        // (Platform functions may change the current OpenGL context, so we save/restore it to make it easier to paste this code elsewhere.
        //  For this specific demo app we could also call glfwMakeContextCurrent(window) directly)
        if (io.ConfigFlags & ImGuiConfigFlags_ViewportsEnable)
        {
            GLFWwindow *backup_current_context = glfwGetCurrentContext();
            ImGui::UpdatePlatformWindows();
            ImGui::RenderPlatformWindowsDefault();
            glfwMakeContextCurrent(backup_current_context);
        }

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}